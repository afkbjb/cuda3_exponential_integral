#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <vector>
#include <cmath>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "ei.h"

static double now_sec() {
    struct timeval tv;
    gettimeofday(&tv, nullptr);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

void printUsage();

int main(int argc, char** argv) {
    // parse args
    unsigned n = 10, m = 10, blk = 256;
    double a = 0.0, b = 10.0;
    bool timing = false, verbose = false;
    bool skipCPU = false, skipGPU = false;
    for (int i = 1; i < argc; ++i) {
        if      (!strcmp(argv[i], "-n")) n       = atoi(argv[++i]);
        else if (!strcmp(argv[i], "-m")) m       = atoi(argv[++i]);
        else if (!strcmp(argv[i], "-a")) a       = atof(argv[++i]);
        else if (!strcmp(argv[i], "-b")) b       = atof(argv[++i]);
        else if (!strcmp(argv[i], "-B")) blk     = atoi(argv[++i]);
        else if (!strcmp(argv[i], "-t")) timing  = true;
        else if (!strcmp(argv[i], "-v")) verbose = true;
        else if (!strcmp(argv[i], "-c")) skipCPU = true;
        else if (!strcmp(argv[i], "-g")) skipGPU = true;
        else if (!strcmp(argv[i], "-h")) { printUsage(); return 0; }
    }

    unsigned total = n * m;
    std::vector<float>  xsF(m),  cpuF(total), gpuF(total);
    std::vector<double> xsD(m), cpuD(total), gpuD(total);
    for (unsigned j = 0; j < m; ++j) {
        double x = a + (j + 1) * (b - a) / m;
        xsF[j] = (float)x;
        xsD[j] = x;
    }

    double cpuFsec = 0.0, cpuDsec = 0.0;
    double gpuFsec = 0.0, gpuDsec = 0.0;

    // --- CPU float + double ---
    if (!skipCPU) {
        double t0 = now_sec();
        for (unsigned o = 0; o < n; ++o)
          for (unsigned j = 0; j < m; ++j)
            cpuF[o*m + j] = exponentialIntegralFloat(o+1, xsF[j]);
        double t1 = now_sec();
        cpuFsec = t1 - t0;

        double t2 = now_sec();
        for (unsigned o = 0; o < n; ++o)
          for (unsigned j = 0; j < m; ++j)
            cpuD[o*m + j] = exponentialIntegralDouble(o+1, xsD[j]);
        double t3 = now_sec();
        cpuDsec = t3 - t2;
    }

    // --- GPU float + double with streams and events ---
    if (!skipGPU) {
        hipFree(0);

        hipStream_t sF, sD;
        hipStreamCreate(&sF);
        hipStreamCreate(&sD);

        hipEvent_t startF, stopF, startD, stopD;
        hipEventCreate(&startF); hipEventCreate(&stopF);
        hipEventCreate(&startD); hipEventCreate(&stopD);

        float  *d_xf, *d_of;
        double *d_xd, *d_od;
        hipMalloc(&d_xf, m     * sizeof(float));
        hipMalloc(&d_of, total * sizeof(float));
        hipMalloc(&d_xd, m     * sizeof(double));
        hipMalloc(&d_od, total * sizeof(double));

        hipEventRecord(startF, sF);
        hipMemcpyAsync(d_xf, xsF.data(), m * sizeof(float),
                        hipMemcpyHostToDevice, sF);
        {
            unsigned grid = (total + blk - 1) / blk;
            expIntFloatKernel<<<grid, blk, 0, sF>>>(n, d_xf, d_of, m, 0, total);
        }
        hipMemcpyAsync(gpuF.data(), d_of, total * sizeof(float),
                        hipMemcpyDeviceToHost, sF);
        hipEventRecord(stopF, sF);

        hipEventRecord(startD, sD);
        hipMemcpyAsync(d_xd, xsD.data(), m * sizeof(double),
                        hipMemcpyHostToDevice, sD);
        {
            unsigned grid = (total + blk - 1) / blk;
            expIntDoubleKernel<<<grid, blk, 0, sD>>>(n, d_xd, d_od, m, 0, total);
        }
        hipMemcpyAsync(gpuD.data(), d_od, total * sizeof(double),
                        hipMemcpyDeviceToHost, sD);
        hipEventRecord(stopD, sD);

        hipStreamSynchronize(sF);
        hipStreamSynchronize(sD);

        float msF = 0.0f, msD = 0.0f;
        hipEventElapsedTime(&msF, startF, stopF);
        hipEventElapsedTime(&msD, startD, stopD);
        gpuFsec = msF * 1e-3;
        gpuDsec = msD * 1e-3;

        hipFree(d_xf); hipFree(d_of);
        hipFree(d_xd); hipFree(d_od);
        hipEventDestroy(startF); hipEventDestroy(stopF);
        hipEventDestroy(startD); hipEventDestroy(stopD);
        hipStreamDestroy(sF); hipStreamDestroy(sD);
    }

    // --- print timing ---
    if (timing) {
        printf("\n==== Timing (blk=%u) ====", blk);
        if (!skipCPU) {
            printf("\nCPU (float) : %.3f ms", cpuFsec * 1e3);
            printf("\nCPU (double): %.3f ms", cpuDsec * 1e3);
        }
        if (!skipGPU) {
            printf("\nGPU (float) : %.3f ms", gpuFsec * 1e3);
            printf("\nGPU (double): %.3f ms", gpuDsec * 1e3);
        }
        if (!skipCPU && !skipGPU) {
            printf("\nSpeedup (float) : %.2fx", cpuFsec / gpuFsec);
            printf("\nSpeedup (double): %.2fx", cpuDsec / gpuDsec);
        }
        printf("\n=================\n");
    }

    // --- verbose compare ---
    if (verbose && !skipCPU && !skipGPU) {
        printf("\n======= Comparison =======\n");
        for (unsigned i = 0; i < n; ++i)
        for (unsigned j = 0; j < m; ++j) {
            unsigned idx = i*m + j;
            double x = a + (j + 1) * (b - a) / m;
            printf("(n=%2u, x=%.1f):\n", i+1, x);
            printf("    float  CPU=%.6g  GPU=%.6g\n", cpuF[idx], gpuF[idx]);
            printf("    double CPU=%.6g  GPU=%.6g\n", cpuD[idx], gpuD[idx]);
        }
        printf("===========================\n");
    }

    return 0;
}

void printUsage () {
    printf("exponentialIntegral program\n");
    printf("by: Jose Mauricio Refojo <refojoj@tcd.ie>\n");
    printf("This program will calculate a number of exponential integrals\n");
    printf("usage:\n");
    printf("exponentialIntegral.out [options]\n");
    printf("      -a   value   : will set the a value of the (a,b) interval in which the samples are taken to value (default: 0.0)\n");
    printf("      -b   value   : will set the b value of the (a,b) interval in which the samples are taken to value (default: 10.0)\n");
    printf("      -c           : will skip the CPU test\n");
    printf("      -g           : will skip the GPU test\n");
    printf("      -h           : will show this usage\n");
    printf("      -i   size    : will set the number of iterations to size (default: 2000000000)\n");
    printf("      -n   size    : will set the n (the order up to which we are calculating the exponential integrals) to size (default: 10)\n");
    printf("      -m   size    : will set the number of samples taken in the (a,b) interval to size (default: 10)\n");
    printf("      -t           : will output the amount of time that it took to generate each norm (default: no)\n");
    printf("      -v           : will activate the verbose mode  (default: no)\n");
    printf("      -B   value   : will set the number of threads per block to value (default: 256)\n");
    printf("     \n");
}
